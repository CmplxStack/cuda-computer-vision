#include "hip/hip_runtime.h"
#include <stdio.h>
#include <unistd.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <opencv2/core/core.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>
#include "../edge-detect/edge_detect.h"
#include "../separable-convolution/separable_convolution.h"
#include "../helper/hip/hip_runtime_api.h"

using namespace cv;

#define TX 16
#define TY 16


void serial_spatial_difference_density_map(double *density_map, int *difference, int width, int height, int horizontal_divisions, int vertical_divisions) {
	int horizontal_block_size = width/horizontal_divisions;
	int vertical_block_size = height/vertical_divisions;
	int block_size = horizontal_block_size * vertical_block_size;
	
	const int scaling_factor = 1000;  // Used to linearly scale density map to units millipixels/pixels^2 (if that makes any sense?)
	
	for (int block_x_index = 0; block_x_index < horizontal_divisions - 1; block_x_index++) {
		for (int block_y_index = 0; block_y_index < vertical_divisions - 1; block_y_index++) {
			int num_differences = 0;
			for (int x = block_x_index * horizontal_block_size; x < (block_x_index + 1) * horizontal_block_size; x++) {
				for (int y = block_y_index * vertical_block_size; y < (block_y_index + 1) * vertical_block_size; y++) {
					if (x > 0 && y > 0 && x < width && y < height && difference[y * width + x] == 255) {
						num_differences++;
					}
				}
			}
			density_map[block_y_index * horizontal_divisions + block_x_index] = scaling_factor*num_differences/(double)block_size;
		}
	}
}

__global__ void spatial_difference_density_map(double *density_map, int *difference, int width, int height, int horizontal_divisions, int vertical_divisions) {
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int i = r * width + c;
	
	int horizontal_block_size = width/horizontal_divisions;
	int vertical_block_size = height/vertical_divisions;
	int block_size = horizontal_block_size * vertical_block_size;
	
	const int scaling_factor = 1000;
	if (difference[i] != 0) {
		density_map[(int)(vertical_divisions*r/(double)height) * horizontal_divisions + (int)(horizontal_divisions*c/(double)width)] += scaling_factor/(double)block_size;
	}
}

__global__ void motion_area_estimate(int *motion_area, double *density_map, int width, int height, int horizontal_divisions, int vertical_divisions, double threshold) {
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int i = r * width + c;
	
	int density_map_index = (int)(vertical_divisions*r/(double)height) * horizontal_divisions + (int)(horizontal_divisions*c/(double)width);

	if (density_map[density_map_index] >= threshold) {
		motion_area[i] = 255;
	} else {
		motion_area[i] = 0;
	}
}

void serial_motion_area_estimate(int *motion_area, double *density_map, int width, int height, int horizontal_divisions, int vertical_divisions, double threshold) {
	int horizontal_block_size = width/horizontal_divisions;
	int vertical_block_size = height/vertical_divisions;
	
	for (int init_i = 0; init_i < width * height; init_i++) {
		motion_area[init_i] = 0;
	}
	for (int i = 0; i < horizontal_divisions * vertical_divisions; i++) {
		if (density_map[i] >= threshold) {
			int r = i/horizontal_divisions;
			int c = i - r*horizontal_divisions;
			for (int x = c * horizontal_block_size; x < (c + 1) * horizontal_block_size; x++) {
				for (int y = r * vertical_block_size; y < (r + 1) * vertical_block_size; y++) {
					motion_area[y*width + x] = 255;
				}
			}
		}
	}
}

__global__ void difference_filter(int *dev_out, int *edges_1, int *edges_2, int width, int height, int threshold) {
    // Note: width should correspond to width of dev_out, edges_1, and edges_2; same for height
	int r = blockIdx.y * blockDim.y + threadIdx.y;
	int c = blockIdx.x * blockDim.x + threadIdx.x;
	int i = r * width + c;

    // Set it to 0 initially
    dev_out[i] = 0;
    int crop_size = 7;
    if (r > crop_size && c > crop_size && r < height - crop_size && c < width - crop_size && edges_1[i] != edges_2[i]) {
        // Set to 255 if there is a pixel mismatch
        dev_out[i] = 255;
        for (int x_apron = -threshold; x_apron <= threshold; x_apron++) {
            for (int y_apron = -threshold; y_apron <= threshold; y_apron++) {
                // Ensure the requested index is within bounds of image
                if (c + x_apron > 0 && r + y_apron > 0 && c + x_apron < width && r + y_apron < height) {
                    // Check if there is a matching pixel in the apron, within the threshold
                    if (edges_1[(r + y_apron) * width + c + x_apron] == edges_2[i]) {
                        // Set it back to 0 if a corresponding pixel exists within the vicinity of the match
                        dev_out[i] = 0;
                    }
                }
            }
        }
    }
    __syncthreads();
}

void serial_difference_filter(int *difference, int *edges_1, int *edges_2, int width, int height, int threshold) {
    for (int x = 0; x < width; x++) {
        for (int y = 0; y < height; y++) {
            difference[y * width + x] = 0;
            if (edges_1[y * width + x] != edges_2[y * width + x]) {
                difference[y * width + x] = 255;
                for (int x_apron = -threshold; x_apron <= threshold; x_apron++) {
                    for (int y_apron = -threshold; y_apron <= threshold; y_apron++) {
                        if (x + x_apron > 0 && y + y_apron > 0 && x + x_apron < width && y + y_apron < height) {
                            if (edges_1[(y + y_apron) * width + x + x_apron] == edges_2[y * width + x]) {
                                difference[y * width + x] = 0;
                            }
                        }
                    }
                }
            }
        }
    }
}

void serial_motion_detect(int *motion_area, int *difference, int *edges_1, int *edges_2, int width, int height, int movement_threshold, int motion_threshold, int horizontal_divisions, int vertical_divisions) {
    double *density_map = (double *)calloc(horizontal_divisions * vertical_divisions, sizeof(double));    
    serial_difference_filter(difference, edges_1, edges_2, width, height, movement_threshold);
	serial_spatial_difference_density_map(density_map, difference, width, height, horizontal_divisions, vertical_divisions);
	serial_motion_area_estimate(motion_area, density_map, width, height, horizontal_divisions, vertical_divisions, motion_threshold);
	free(density_map);
}

void motion_detect(int *motion_area, int *difference, int *edges_1, int *edges_2, int width, int height, int movement_threshold, int motion_threshold, int horizontal_divisions, int vertical_divisions) {
	// Note: movement_threshold refers to the pixel apron around which the difference filter attempts to look for differences.
	// Higher movement_threshold == more leniency in how much camera shake is tolerated
	// motion_threshold refers to the minimum spatial pixel difference density required for a particular segment of the difference to be registered as motion.
	// Lower motion_threshold == more sensitive in picking up motion
	
    // Allocate space on device
    int *dev_edges_1, *dev_edges_2, *dev_difference, *dev_motion_area;
    checkCudaErrors(hipMalloc(&dev_edges_1, width*height*sizeof(int)));
    checkCudaErrors(hipMalloc(&dev_edges_2, width*height*sizeof(int)));
    checkCudaErrors(hipMalloc(&dev_difference, width*height*sizeof(int)));
    checkCudaErrors(hipMalloc(&dev_motion_area, width*height*sizeof(int)));
    
    double *density_map = (double *)calloc(horizontal_divisions * vertical_divisions, sizeof(double));
    double *dev_density;
    checkCudaErrors(hipMalloc(&dev_density, horizontal_divisions*vertical_divisions*sizeof(double)));
    checkCudaErrors(hipMemcpy(dev_density, density_map, horizontal_divisions * vertical_divisions * sizeof(double), hipMemcpyHostToDevice));

    // Copy host arrays to device
    checkCudaErrors(hipMemcpy(dev_edges_1, edges_1, width*height*sizeof(int), hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(dev_edges_2, edges_2, width*height*sizeof(int), hipMemcpyHostToDevice));

//    int *serial_output = (int *)malloc(width * height * sizeof(int));
//    serial_difference_filter(difference, edges_1, edges_2, width, height, movement_threshold);

    // Initialize grid
	dim3 block_size(TX, TY);
	int bx = width/block_size.x;
	int by = height/block_size.y;
	dim3 grid_size = dim3(bx, by);

	// Difference filter
    difference_filter<<<grid_size, block_size>>>(dev_difference, dev_edges_1, dev_edges_2, width, height, movement_threshold);
    checkCudaErrors(hipMemcpy(difference, dev_difference, width * height * sizeof(int), hipMemcpyDeviceToHost));

    // Determine spatial density map
	spatial_difference_density_map<<<grid_size, block_size>>>(dev_density, dev_difference, width, height, horizontal_divisions, vertical_divisions);
//	serial_spatial_difference_density_map(density_map, difference, width, height, horizontal_divisions, vertical_divisions);
	
	// Estimate motion area
//	serial_motion_area_estimate(motion_area, density_map, width, height, horizontal_divisions, vertical_divisions, motion_threshold);
	motion_area_estimate<<<grid_size, block_size>>>(dev_motion_area, dev_density, width, height, horizontal_divisions, vertical_divisions, motion_threshold);
	checkCudaErrors(hipMemcpy(motion_area, dev_motion_area, width * height * sizeof(int), hipMemcpyDeviceToHost));
	
	// Responsible programmer
	checkCudaErrors(hipFree(dev_density));
	checkCudaErrors(hipFree(dev_motion_area));
	checkCudaErrors(hipFree(dev_difference));
	checkCudaErrors(hipFree(dev_edges_1));
	checkCudaErrors(hipFree(dev_edges_2));
	
	// Free allocated host memory
	free(density_map);
}