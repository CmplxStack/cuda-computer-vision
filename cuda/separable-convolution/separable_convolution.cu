#include "hip/hip_runtime.h"
#include <stdio.h>
#include <sys/time.h>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <iostream>

using namespace cv;


__global__ void horizontal_convolve(int *d_out, int *x, int *h, int x_width, int x_height, int h_width, int h_height) {
    const int r = blockIdx.x;
    const int c = threadIdx.x;
    const int i = r * blockDim.x + c;

    int sum = 0;
    for (int j = 0; j < h_width; j++) {
        int p = x_width*r + c - j;
        if (c - j >= 0 && c - j < x_width) {
            sum += h[j] * x[p];
        }
    }
    d_out[i] = sum;
    __syncthreads();
}

__global__ void vertical_convolve(int *d_out, int *x, int *h, int x_width, int x_height, int h_width, int h_height, double constant_scalar) {
    const int r = blockIdx.x;
    const int c = threadIdx.x;
    const int i = r * blockDim.x + c;

    int sum = 0;
    for (int j = 0; j < h_height; j++) {
        int p = x_width*(r - j) + c;
        if (r - j >= 0 && r - j < x_height) {
            sum += h[j] * x[p];
        }
    }
    d_out[i] = (int)(constant_scalar * (double)sum);
    __syncthreads();
}

double serial_separable_convolve(int *out, int *x, int *horizontal_filter, int *vertical_filter, int x_width, int x_height, int horizontal_filter_width, int vertical_filter_height, double constant_scalar) {
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);
    int *horizontal_out = (int *)malloc((x_width + horizontal_filter_width - 1) * x_height * sizeof(int));
    for (int m = 0; m < x_height; m++) {
        for (int n = 0; n < x_width + horizontal_filter_width - 1; n++) {
            int h_sum = 0;
            for (int j = 0; j < horizontal_filter_width; j++) {
                if (n - j >= 0 && n - j < x_width) {
                    h_sum += x[m * x_width + n - j] * horizontal_filter[j];
                }
            }
            horizontal_out[m * (x_width + horizontal_filter_width - 1) + n] = h_sum;
        }
    }
    for (int v_m = 0; v_m < x_height + vertical_filter_height - 1; v_m++) {
        for (int v_n = 0; v_n < x_width + horizontal_filter_width - 1; v_n++) {
            int v_sum = 0;
            for (int i = 0; i < vertical_filter_height; i++) {
                if (v_m - i >= 0 && v_m - i < x_height) {
                    v_sum += horizontal_out[(v_m - i) * (x_width + horizontal_filter_width - 1) + v_n] * vertical_filter[i];
                }
            }
            out[v_m * (x_width + horizontal_filter_width - 1) + v_n] = (int)(constant_scalar * (double)v_sum);
        }
    }
    gettimeofday(&tv2, NULL);
    double time_spent = (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
    printf ("Serial separated convolution execution time: %f seconds\n", time_spent);
    return time_spent;
}

double serial_convolve(int *out, int *x, int *h, int x_width, int x_height, int h_width, int h_height) {
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);
    for (int m = 0; m < x_height + h_height - 1; m++) {
        for (int n = 0; n < x_width + h_width - 1; n++) {
            int sum = 0;
            for (int i = 0; i < h_height; i++) {
                for (int j = 0; j < h_width; j++) {
                    if (m - i >= 0 && m - i < x_height && n - j >= 0 && n - j < x_width) {
                        sum += h[i * h_width + j] * x[(m - i) * x_width + n - j];
                    }
                }
            }
            out[m * (x_width + h_width - 1) + n] = sum;
        }
    }
    gettimeofday(&tv2, NULL);
    double time_spent = (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
    printf ("Serial naive convolution execution time: %f seconds\n", time_spent);
    return time_spent;
}

void separable_convolve(int *output, int *x, int x_width, int x_height, int *horizontal_filter, int *vertical_filter, int kernel_size, double constant_scalar) {
    // Specify lengths of filters and input
    int horizontal_filter_width = kernel_size;
    int vertical_filter_height = kernel_size;

    // Allocate space for host and device arrays
    int *dev_horizontal_out, *dev_vertical_out;  // Results of the horizontal and vertical convolutions on the input array
    int *dev_horizontal_filter, *dev_vertical_filter, *dev_x;  // Horizontal filter, vertical filter, and input array
    //int *output = (int *)malloc((x_width * x_height + horizontal_filter_width + vertical_filter_height) * sizeof(int));
    //int *serial_output = (int *)malloc((x_width * x_height + horizontal_filter_width + vertical_filter_height) * sizeof(int));
    static int serial_output[10000000];

    // Horizontal filter, followed by vertical filter
    int horizontal_convolution_width = x_width + horizontal_filter_width - 1;
    int horizontal_convolution_height = x_height;
    int vertical_convolution_width = horizontal_convolution_width;
    int vertical_convolution_height = horizontal_convolution_height + vertical_filter_height - 1;

    // Allocate space for horizontal result, vertical result, horizontal filter, vertical filter, and input
    hipMalloc(&dev_horizontal_out, horizontal_convolution_width*horizontal_convolution_height*sizeof(int));
    hipMalloc(&dev_vertical_out, vertical_convolution_width*vertical_convolution_height*sizeof(int));
    hipMalloc(&dev_horizontal_filter, horizontal_filter_width*sizeof(int));
    hipMalloc(&dev_vertical_filter, vertical_filter_height*sizeof(int));
    hipMalloc(&dev_x, x_width*x_height*sizeof(int));

    // Copy host arrays to device
    hipMemcpy(dev_horizontal_filter, horizontal_filter, horizontal_filter_width*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_vertical_filter, vertical_filter, vertical_filter_height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, x_width*x_height*sizeof(int), hipMemcpyHostToDevice);

    // Start with a serial convolution for comparison
    // double serial_computation_time = serial_convolve(serial_output, x, filter, x_width, x_height, horizontal_filter_width, vertical_filter_height);
    double serial_computation_time = serial_separable_convolve(serial_output, x, horizontal_filter, vertical_filter, x_width, x_height, horizontal_filter_width, vertical_filter_height, constant_scalar);

    // Start a timer and do the two convolutions
    struct timeval tv1, tv2;
    gettimeofday(&tv1, NULL);
    horizontal_convolve<<<horizontal_convolution_height, horizontal_convolution_width>>>(dev_horizontal_out, dev_x, dev_horizontal_filter, x_width, x_height, horizontal_filter_width, 1);
    vertical_convolve<<<vertical_convolution_height, vertical_convolution_width>>>(dev_vertical_out, dev_horizontal_out, dev_vertical_filter, horizontal_convolution_width, horizontal_convolution_height, 1, vertical_filter_height, constant_scalar);

    // Copy result data from device to host
    hipMemcpy(output, dev_vertical_out, vertical_convolution_width*vertical_convolution_height*sizeof(int), hipMemcpyDeviceToHost);

    // Responsible programmer
    hipFree(dev_vertical_out);
    hipFree(dev_horizontal_out);
    hipFree(dev_horizontal_filter);
    hipFree(dev_vertical_filter);
    hipFree(dev_x);

    // Parallel computation time
    gettimeofday(&tv2, NULL);
    double parallel_computation_time = (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec);
    printf("Parallel convolution execution time: %f seconds\n", parallel_computation_time);

    // Error checking
    for (int i = 0; i < vertical_convolution_width*vertical_convolution_height; i++) {
        if (serial_output[i] != output[i]) {
            // printf("Error! serial and parallel computation results are inconsistent: %d, %d\n", serial_output[i], output[i]);
        }
    }
    printf("Estimated parallelization speedup: %f\n", serial_computation_time/parallel_computation_time);
}

int main() {
    // Load external image into array
    Mat image = imread("nvidia_1000_1000.jpg", 0);
    int *x = (int *)malloc(image.cols * image.rows * sizeof(int));
    int *out = (int *)malloc(100000000 * sizeof(int));
    for (int i = 0; i < image.rows; i++) {
        for (int j = 0; j < image.cols; j++) {
            x[i * image.rows + j] = image.at<uchar>(i, j);
        }
    }

    // Gaussian filter
    int horizontal_filter[3] = {1, 2, 1};
    int vertical_filter[3] = {1, 2, 1};
    int kernel_size = 3;
    double constant_scalar = 1.0/16.0;
    separable_convolve(out, x, image.cols, image.rows, horizontal_filter, vertical_filter, kernel_size, constant_scalar);

    // Write to disk
    Mat m(image.rows + kernel_size - 1, image.cols + kernel_size - 1, CV_32SC1, out);
    imwrite("temp.jpg", m);

    return 0;
}
