
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <sys/time.h>

__global__ void horizontal_convolve(int *d_out, int *x, int *h, int x_width, int x_height, int h_width, int h_height) {
    const int r = blockIdx.x;
    const int c = threadIdx.x;
    const int i = r * blockDim.x + c;

    int sum = 0;
    for (int j = 0; j < h_width; j++) {
        int p = x_width*r + c - j;
        if (c - j >= 0 && c - j < h_width) {
            sum += h[j] * x[p];
        }
    }
    d_out[i] = sum;
    __syncthreads();
}

__global__ void vertical_convolve(int *d_out, int *x, int *h, int x_width, int x_height, int h_width, int h_height) {
    const int r = blockIdx.x;
    const int c = threadIdx.x;
    const int i = r * blockDim.x + c;

    int sum = 0;
    for (int j = 0; j < x_height; j++) {
        int p = h_width*(r - j) + c;
        if (r - j >= 0 && r - j < h_height) {
            sum += x[j] * h[p];
        }
    }
    d_out[i] = sum;
    __syncthreads();
}

void serial_convolve(int *out, int *x, int *h, int x_width, int x_height, int h_width, int h_height) {
    struct timeval  tv1, tv2;
    gettimeofday(&tv1, NULL);
    for (int m = 0; m < x_height + h_height - 1; m++) {
        for (int n = 0; n < x_width + h_width - 1; n++) {
            int sum = 0;
            for (int i = 0; i < x_height; i++) {
                for (int j = 0; j < x_width; j++) {
                    if (m - i >= 0 && m - i < h_height && n - j >= 0 && n - j < h_width) {
                        sum += x[i * x_width + j] * h[(m - i) * h_width + n - j];
                    }
                }
            }
            out[m * (x_width + h_width - 1) + n] = sum;
        }
    }
    gettimeofday(&tv2, NULL);
    printf ("Serial convolution execution time: %f seconds\n", (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec));
}

void separable_convolve() {
    int *dev_horizontal_out, *dev_vertical_out;  // Results of the horizontal and vertical convolutions on the input array
    int *dev_horizontal_filter, *dev_vertical_filter, *dev_x;  // Horizontal filter, vertical filter, and input array
    int output[100];

    int horizontal_filter_width = 5;
    int vertical_filter_height = 5;
    int x_width = 2, x_height = 2;

    // Horizontal filter, followed by vertical filter
    int horizontal_convolution_width = x_width + horizontal_filter_width - 1;
    int horizontal_convolution_height = x_height;
    int vertical_convolution_width = horizontal_convolution_width;
    int vertical_convolution_height = horizontal_convolution_height + vertical_filter_height - 1;

    // Allocate space for horizontal result, vertical result, horizontal filter, vertical filter, and input
    hipMalloc(&dev_horizontal_out, horizontal_convolution_width*horizontal_convolution_height*sizeof(int));
    hipMalloc(&dev_vertical_out, vertical_convolution_width*vertical_convolution_height*sizeof(int));
    hipMalloc(&dev_horizontal_filter, horizontal_filter_width*sizeof(int));
    hipMalloc(&dev_vertical_filter, vertical_filter_height*sizeof(int));
    hipMalloc(&dev_x, x_width*x_height*sizeof(int));

    // Load host data
    int horizontal_filter[5] = {1, 2, 3, 4, 5};
    int vertical_filter[5] = {6, 7, 8, 9, 10};
    int x[4] = {1, 2, 3, 4};

    hipMemcpy(dev_horizontal_filter, horizontal_filter, horizontal_filter_width*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_vertical_filter, vertical_filter, vertical_filter_height*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_x, x, x_width*x_height*sizeof(int), hipMemcpyHostToDevice);

    struct timeval  tv1, tv2;
    gettimeofday(&tv1, NULL);
    horizontal_convolve<<<horizontal_convolution_height, horizontal_convolution_width>>>(dev_horizontal_out, dev_x, dev_horizontal_filter, x_width, x_height, horizontal_filter_width, 1);
    //vertical_convolve<<<vertical_convolution_height, vertical_convolution_width>>>(dev_vertical_out, dev_horizontal_out, dev_vertical_filter, horizontal_convolution_width, horizontal_convolution_height, 1, vertical_filter_height);

    //cudaMemcpy(output, dev_vertical_out, vertical_convolution_width*vertical_convolution_height*sizeof(int), cudaMemcpyDeviceToHost);
    hipMemcpy(output, dev_horizontal_out, horizontal_convolution_width*horizontal_convolution_height*sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_vertical_out);
    hipFree(dev_horizontal_out);
    hipFree(dev_horizontal_filter);
    hipFree(dev_vertical_filter);
    hipFree(dev_x);

    for (int i = 0; i < horizontal_convolution_width*horizontal_convolution_height; i++) {
        printf("i %d, output %d\n", i, output[i]);
    }

    gettimeofday(&tv2, NULL);
    printf ("Parallel convolution execution time: %f seconds\n", (double) (tv2.tv_usec - tv1.tv_usec) / 1000000 + (double) (tv2.tv_sec - tv1.tv_sec));
}

int main() {
    separable_convolve();
    return 0;
}
